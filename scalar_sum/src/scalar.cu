
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

const int N = 1;

__global__
void add(int *a, int *b, int *c){
int tID = threadIdx.x;
if(tID < N) *c=*a+*b;
if(tID == N) *c=*a-*b;
}

int main( void ){
int a,b,c;
int *dev_a,*dev_b,*dev_c;
int size = sizeof(int);

a=3;b=2;c=0;

hipMalloc( (void **)&dev_a, size );
hipMalloc( (void **)&dev_b, size );
hipMalloc( (void **)&dev_c, size );

hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice ); 
hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice ); 


add<<<1, 1>>>(dev_a, dev_b, dev_c);
hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
std::cout << "ThreadIDx= " << "0" << std::endl; 
std::cout << " a=" << a << " b=" << b << " a+b=" << c << std::endl;

add<<<1, 2>>>(dev_a, dev_b, dev_c);
hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);
std::cout << "ThreadIDx= " << "1" << std::endl; 
std::cout << " a=" << a << " b=" << b << " a-b=" << c << std::endl;
 

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);

//return EXIT_SUCCESS;
return 0;
}
